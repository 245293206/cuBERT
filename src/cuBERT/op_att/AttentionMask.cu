#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/system/cuda/execution_policy.h>

#include "AttentionMask.h"

namespace cuBERT {
    template<typename T>
    __host__ void _not(const int8_t *in,
                       T *out,
                       const int N,
                       void *stream) {
        thrust::device_ptr<const int8_t> in_ptr(in);
        thrust::device_ptr<T> out_ptr(out);
        thrust::transform(thrust::cuda::par.on((hipStream_t) stream), in_ptr, in_ptr + N, out_ptr, thrust::logical_not<const int8_t>());
    }

    template
    __host__ void _not<float>(const int8_t *in,
                              float *out,
                              const int N,
                              void *stream);

    template
    __host__ void _not<half >(const int8_t *in,
                              half *out,
                              const int N,
                              void *stream);
}
