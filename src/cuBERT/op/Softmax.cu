#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/system/cuda/execution_policy.h>

#include <float.h>

#include "math.h"
#include "hipcub/hipcub.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "Softmax.h"

namespace cuBERT {

    template <typename T>
    struct exp_functor {
        __device__ T operator()(const T& x) const {
            return T(__expf((float) x));
        }
    };

    template <typename T>
    __global__ void kernel_max_cub(const T *__restrict__ in,
                                   const int batch_size,
                                   const int channel,
                                   T *max_out) {
        __shared__ typename hipcub::BlockReduce<float, 128>::TempStorage temp_storage;
        for (int i = blockIdx.x; i < batch_size; i += gridDim.x) {
            float val = -FLT_MAX;
            for (int j = threadIdx.x; j < channel; j += blockDim.x) {
#if __CUDA_ARCH__ >= 350
                val = CUB_MAX((float) __ldg(in + i * channel + j), val);
#else
                val = CUB_MAX((float) in[i * channel + j], val);
#endif
            }
            val = hipcub::BlockReduce<float, 128>(temp_storage).Reduce(val, hipcub::Max());
            if (threadIdx.x == 0) {
                max_out[i] = val;
            }
            __syncthreads();
        }
    }

    template <typename T>
    __global__ void kernel_substract(const T *__restrict__ in, 
                                     T *out, 
                                     const int batch_size, 
                                     const int channel, 
                                     const T *__restrict__ max_in) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= batch_size * channel) {
            return;
        }

        int batch_idx = idx / channel;
#if __CUDA_ARCH__ >= 350
        out[idx] = (float) __ldg(in + idx) - (float) __ldg(max_in + batch_idx);
#else
        out[idx] = (float) in[idx] - (float) max_in[batch_idx];
#endif
    }

    template <typename T>
    __global__ void kernel_sum_cub(const T *__restrict__ in,
                                   const int batch_size,
                                   const int channel,
                                   T *sum_out) {
        __shared__ typename hipcub::BlockReduce<float , 128>::TempStorage s_storage;
        for (int i = blockIdx.x; i < batch_size; i += gridDim.x) {
            float s_val = 0.f;
            for (int j = threadIdx.x; j < channel; j += blockDim.x) {
#if __CUDA_ARCH__ >= 350
                s_val += (float) __ldg(in + i * channel + j);
#else
                s_val += (float) in[i * channel + j];
#endif
            }
            s_val = hipcub::BlockReduce<float, 128>(s_storage).Sum(s_val);
            if (threadIdx.x == 0) {
                sum_out[i] = s_val;
            }
            __syncthreads();
        }
    }

    template <typename T>
    __global__ void kernel_scale_(T *inout, const int batch_size, const int channel, T *sum_in) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= batch_size * channel) {
            return;
        }

        int batch_idx = idx / channel;
#if __CUDA_ARCH__ >= 350
        inout[idx] = (float) __ldg(inout + idx) / (float) __ldg(sum_in + batch_idx);
#else
        inout[idx] = (float) inout[idx] / (float) sum_in[batch_idx];
#endif
    }

    template <typename T>
    __host__ void softmax_(T *in, T *out, const int batch_size, const int channel, T *sum_gpu, void* stream) {
        const int all_blocks = (batch_size * channel + 127) / 128;

        kernel_max_cub<T> <<<batch_size, 128, 0, (hipStream_t) stream>>> (in, batch_size, channel, sum_gpu);
        kernel_substract<T> <<<all_blocks, 128, 0, (hipStream_t) stream>>> (in, out, batch_size, channel, sum_gpu);

        thrust::device_ptr<T> dev_ptr(out);
        thrust::transform(thrust::cuda::par.on((hipStream_t) stream), dev_ptr, dev_ptr + batch_size * channel, dev_ptr, exp_functor<T>());

        kernel_sum_cub<T> <<<batch_size, 128, 0, (hipStream_t) stream>>> (out, batch_size, channel, sum_gpu);
        kernel_scale_<T> <<<all_blocks, 128, 0, (hipStream_t) stream>>> (out, batch_size, channel, sum_gpu);
    }

    template
    __host__ void softmax_<float>(float *in, float *out, const int batch_size, const int channel, float *sum_gpu, void *stream);

    template
    __host__ void softmax_<half>(half *in, half *out, const int batch_size, const int channel, half *sum_gpu, void *stream);
}
