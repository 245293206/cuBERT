#include "hip/hip_runtime.h"
#include "math.h"
#include <hip/hip_runtime.h>

namespace cuBERT {
    __global__ void kernel_layer_norm_(float *inout,
                                       const int batch_size,
                                       const int channel,
                                       const float *__restrict__ beta,
                                       const float *__restrict__ gamma) {
        int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (batch_idx >= batch_size) {
            return;
        }

        // channel data: [batch_idx * channel, (batch_idx + 1) * channel)
        float mean = 0;
        float var = 0;
#pragma unroll
        for (int i = batch_idx * channel; i < (batch_idx + 1) * channel; ++i) {
            float t = __ldg(inout + i);
            mean += t;
            var += t * t;
        }
        mean = mean / channel;
        var = var / channel - mean * mean;

        // 1 / sqrt(var)
        var = rsqrtf(var + 1e-12);

#pragma unroll
        for (int i = 0; i < channel; ++i) {
            int j = batch_idx * channel + i;
            inout[j] = __ldg(beta + i) + __ldg(gamma + i) * var * (__ldg(inout + j) - mean);
        }
    }

    __host__ void layer_norm_(float *inout,
                              const int batch_size,
                              const int channel,
                              float *beta,
                              float *gamma,
                              hipStream_t stream) {
        const int blocks = (batch_size + 127) / 128;
        kernel_layer_norm_ << < blocks, 128, 0, stream >> > (inout, batch_size, channel, beta, gamma);
    }

    __global__ void kernel_layer_norm_(float *in,
                                       float *inout,
                                       const int batch_size,
                                       const int channel,
                                       const float *__restrict__ beta,
                                       const float *__restrict__ gamma) {
        int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (batch_idx >= batch_size) {
            return;
        }

        // channel data: [batch_idx * channel, (batch_idx + 1) * channel)
        float mean = 0;
        float var = 0;
#pragma unroll
        for (int i = batch_idx * channel; i < (batch_idx + 1) * channel; ++i) {
            float t = __ldg(inout + i) + __ldg(in + i);
            mean += t;
            var += t * t;
        }
        mean = mean / channel;
        var = var / channel - mean * mean;

        // 1 / sqrt(var)
        var = rsqrtf(var + 1e-12);

#pragma unroll
        for (int i = 0; i < channel; ++i) {
            int j = batch_idx * channel + i;
            inout[j] = __ldg(beta + i) + __ldg(gamma + i) * var * (__ldg(inout + j) + __ldg(in + j) - mean);
        }
    }

    __host__ void layer_norm_(float *in,
                              float *inout,
                              const int batch_size,
                              const int channel,
                              float *beta,
                              float *gamma,
                              hipStream_t stream) {
        const int blocks = (batch_size + 127) / 128;
        kernel_layer_norm_ << < blocks, 128, 0, stream >> > (in, inout, batch_size, channel, beta, gamma);
    }
}
