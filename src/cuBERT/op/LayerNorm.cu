#include "hip/hip_runtime.h"
#include "math.h"
#include "hipcub/hipcub.hpp"
#include <hip/hip_runtime.h>

namespace cuBERT {
    __global__ void kernel_layer_norm_(float *inout,
                                       const int batch_size,
                                       const int channel,
                                       const float *__restrict__ beta,
                                       const float *__restrict__ gamma) {
        int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (batch_idx >= batch_size) {
            return;
        }

        // channel data: [batch_idx * channel, (batch_idx + 1) * channel)
        float mean = 0;
        float var = 0;
#pragma unroll
        for (int i = batch_idx * channel; i < (batch_idx + 1) * channel; ++i) {
            float t = __ldg(inout + i);
            mean += t;
            var += t * t;
        }
        mean = mean / channel;
        var = var / channel - mean * mean;

        // 1 / sqrt(var)
        var = rsqrtf(var + 1e-12);

#pragma unroll
        for (int i = 0; i < channel; ++i) {
            int j = batch_idx * channel + i;
            inout[j] = __ldg(beta + i) + __ldg(gamma + i) * var * (__ldg(inout + j) - mean);
        }
    }

    __host__ void layer_norm_(float *inout,
                              const int batch_size,
                              const int channel,
                              float *beta,
                              float *gamma,
                              void* stream) {
        const int blocks = (batch_size + 127) / 128;
        kernel_layer_norm_ << < blocks, 128, 0, (hipStream_t) stream >> > (inout, batch_size, channel, beta, gamma);
    }

    __global__ void kernel_momentum_cub(const float *__restrict__ in,
                                        const float *__restrict__ inout,
                                        const int batch_size,
                                        const int channel,
                                        float *mean_out,
                                        float *var_out) {
        __shared__ typename hipcub::BlockReduce<float, 128>::TempStorage m_storage;
        __shared__ typename hipcub::BlockReduce<float, 128>::TempStorage v_storage;
        const float scale = 1.f / channel;
        for (int i = blockIdx.x; i < batch_size; i += gridDim.x) {
            float m_val = 0;
            float v_val = 0;
            for (int j = threadIdx.x; j < channel; j += blockDim.x) {
                const int X_index = i * channel + j;
                const float t = __ldg(in + X_index) + __ldg(inout + X_index);
                m_val += t;
                v_val += t * t;
            }
            m_val = hipcub::BlockReduce<float, 128>(m_storage).Sum(m_val);
            v_val = hipcub::BlockReduce<float, 128>(v_storage).Sum(v_val);
            if (threadIdx.x == 0) {
                const float mu = m_val * scale;
                mean_out[i] = mu;
                var_out[i] = v_val * scale - mu * mu;
            }
            __syncthreads();
        }
    }

    __global__ void kernel_batchnorm_(const float *__restrict__ in,
                                      float *inout,
                                      const int batch_size,
                                      const int channel,
                                      const float *__restrict__ mean_in,
                                      const float *__restrict__ var_in,
                                      const float *__restrict__ beta,
                                      const float *__restrict__ gamma) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= batch_size * channel) {
            return;
        }

        int batch_idx = idx / channel;
        int channel_idx = idx % channel;

        float mean = __ldg(mean_in + batch_idx);
        float var = __ldg(var_in + batch_idx);

        // 1 / sqrt(var)
        var = rsqrtf(var + 1e-12);

        inout[idx] = __ldg(beta + channel_idx) +
                __ldg(gamma + channel_idx) * var * (__ldg(inout + idx) + __ldg(in + idx) - mean);
    }

    __host__ void layer_norm_(float *in,
                              float *inout,
                              const int batch_size,
                              const int channel,
                              float *mean_gpu,
                              float *var_gpu,
                              float *beta,
                              float *gamma,
                              void* stream) {
        kernel_momentum_cub <<<batch_size, 128, 0, (hipStream_t) stream>>> (in, inout, batch_size, channel, mean_gpu, var_gpu);

        const int all_blocks = (batch_size * channel + 127) / 128;
        kernel_batchnorm_ <<<all_blocks, 128, 0, (hipStream_t) stream>>> (in, inout, batch_size, channel, mean_gpu, var_gpu, beta, gamma);
    }
}
