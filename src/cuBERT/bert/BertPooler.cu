#include "hip/hip_runtime.h"
#include "math.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/system/cuda/execution_policy.h>

namespace cuBERT {
    struct tanh_functor {
        __device__ float operator()(const float& x) const {
            return tanhf(x);
        }
    };

    __host__ void tanh_(float *inout,
                        const int N,
                        hipStream_t stream) {
        thrust::device_ptr<float> dev_ptr(inout);
        thrust::transform(thrust::cuda::par.on(stream), dev_ptr, dev_ptr + N, dev_ptr, tanh_functor());
    }
}
